#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;




__global__ void Draw(char *frame) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H && x < W) {
		char c;
		//char drawing[] = "::::::::::::::::::::::::::::::::::::::: :                                     : :                                     : :                                     : :                                     : :                 ####          <|    : :               ######           |    : :             ########           |    : :           ##########           |    : :         ############           |    : :       ##############           #    : ::::::::::::::::::::::::::::::::::::::: ";
		char drawing[] = "   MARIO             WORLD    TIME      001126    $x25     1-1      127                                                                                              ####          <|                   ######           |                 ########           |               ##########           |             ############           |           ##############           #    ";
		if (x == W-1) {
			c = y == H-1 ? '\0' : '\n';
		} else if (y == 0 || y == H-1 || x == 0 || x == W-2) {
			c = ':';
		} else {
			c = drawing[(y-1)*(W-3)+(x-1)];


			//c = ' ';
		}
		frame[y*W+x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;
	return 0;
}